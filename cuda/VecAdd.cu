
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void VecAdd(double* A, double* B, double* C, int N) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] + B[i];
}

#define N 10000

int main() {
  double a[N];
  double b[N];
  double c[N];

  int i;
  for (i = 0; i < N; ++i) {
    a[i] = 3.0*(double)i - 11.4;
    b[i] = 42.0 / ((double)i);
  }

  size_t s = N * sizeof(double);

  double* da;
  hipMalloc((void**)&da, s);
  double* db;
  hipMalloc((void**)&db, s);
  double* dc;
  hipMalloc((void**)&dc, s);

  hipMemcpy(da, a, s, hipMemcpyHostToDevice);
  hipMemcpy(db, b, s, hipMemcpyHostToDevice);

  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  VecAdd<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, N);

  hipMemcpy(c, dc, s, hipMemcpyDeviceToHost);
  
  for (i = 0; i < 10000; ++i) {
    printf("%.3lf + %.3lf = %.3lf\n", a[i], b[i], c[i]);
  }
  
  hipFree(da);
  hipFree(db);
  hipFree(dc);

  return 0;
}
