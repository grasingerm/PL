#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 1000

__global__ void MatAdd(float A[N][N], float B[N][N], float C[N][N]) {
  int i = threadIdx.x;
  int j = threadIdx.y;
  C[i][j] = A[i][j] + B[i][j];
}

int main() {
  float a[N][N];
  float b[N][N];
  float c[N][N];

  int i, j;
  for (i = 0; i < N; ++i) {
    for (j = 0; j < N; ++j) {
      a[i][j] = 3.0 * i - 7.0 * j * j;
      b[i][j] = (double) i / (double) j;
    }
  }

  size_t size = N * N * sizeof(float);

  float* d_A;
  hipMalloc(&d_A, size);
  float* d_B;
  hipMalloc(&d_B, size);
  float* d_C;
  hipMalloc(&d_C, size);

  cubaMemcpy(d_A, a, size, hipMemcpyHostToDevice);
  cubaMemcpy(d_B, b, size, hipMemcpyHostToDevice);

  int numBlocks = 1;
  dim3 threadsPerBlock(N, N);
  MatAdd<<<numBlocks, threadsPerBlock>>>(a, b, c);

  hipMemcpy(c, d_C, size, hipMemcpyDeviceToHost);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  for (i = 0; i < N; ++i) {
    for (j = 0; j < N; ++j) {
      printf("%.3lf + %.3lf = %.3lf\n", a[i][j], b[i][j], c[i][j]);
    }
  }

  return 0;
}
